#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "error.h"


// Prints "Hello world!" from GPU.
// A kernel function in CUDA is defined with __global__. NVCC picks it up and
// generates (intermediate) GPU code for this function. It also generates a
// placeholder (CPU code) that can be trigger the execution of this kernel
// through CUDA runtime. Its return type is always "void".
// 1. printf is managed by CUDA driver (how?).
__global__ void kernel() {
  printf("GPU: Hello world!\n"); // 1
}


// Prints "Hello world!" from GPU & CPU.
// A kernel is called with "kernel<<<blocks, threads>>>(arguments...)" syntax.
// Each execution of kernel is called a thread. A number of threads are grouped
// into thread blocks. All thread blocks of a kernel call are grouped into a
// grid. Threads within a block can communicate and synchronize with each
// other, but blocks execute independently of each other (though they can still
// communicate through global GPU memory).
// 1. Kernel is called with 12 "threads".
// 2. Waits for GPU to finish executing the kernel.
// 3. CPU prints after GPU is already done.
int main() {
  kernel<<<1, 12>>>();          // 1
  TRY(hipDeviceSynchronize()); // 2
  printf("CPU: Hello world!");  // 3
  return 0;
}
